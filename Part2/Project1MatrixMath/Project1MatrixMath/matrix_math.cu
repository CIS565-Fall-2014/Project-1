/* 
 * MICHAEL B LI
 * CIS 565 Project-1
 * 
 * "Hello world" made possible with support from http://www.brainlings.com/2011/11/hello-world-in-cuda/
 * 
 * 
 */


#include <hip/hip_runtime.h>
#include <iostream>

#define MAT_WIDTH 5

// forward declare CPU helper method to display results
void printResults(float*, char*);
// and the serial versions of stuff
void cpu_mat_add (float*, float*, float*);
void cpu_mat_sub (float*, float*, float*);
void cpu_mat_mult (float*, float*, float*);

// kernels to run on GPU
__global__ void mat_add(float* Md, float* Nd, float* Pd) {
	int tx = threadIdx.x; // COLUMN counter: incr 1 -> add 1
	int ty = threadIdx.y; // ROW counter:    incr 1 -> add 5 (or whatever MAT_WIDTH is)

	int index = ty * MAT_WIDTH + tx;

	Pd[index] = Md[index] + Nd[index];
}

__global__ void mat_sub(float* Md, float* Nd, float* Pd) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int index = ty * MAT_WIDTH + tx;

	Pd[index] = Md[index] - Nd[index];
}

__global__ void mat_mult(float* Md, float* Nd, float* Pd) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float runningSum = 0.f; // to be used for calculation of the dot product

	// M: move along a row, increment COLUMN counter
	// N: move along a column, increment ROW counter
	for (int k = 0; k < MAT_WIDTH; k++) {
		runningSum += Md[ty*MAT_WIDTH + k] * Nd[tx + k*MAT_WIDTH];
	}

	Pd[ty * MAT_WIDTH + tx] = runningSum;
}


int main(int argc, char** argv) {

	//std::cout << "Hello there!" << std::endl;

	//std::cin.ignore();

	//return 0;

	//#################################################################################


	// CPU float array
	// I will just be using a 1D array to hold the floats.
	// MAT_WIDTH is defined to be 5 above; change it to test with something else. All matrices assumed to be SQUARE.

	int size = MAT_WIDTH * MAT_WIDTH * sizeof(float); // how much memory needed

	// initialize 2 matrices on the CPU
	float* M = new float[MAT_WIDTH*MAT_WIDTH]();
	float* N = new float[MAT_WIDTH*MAT_WIDTH]();
	for (int i = 0; i < MAT_WIDTH*MAT_WIDTH; i++) {
		M[i] = float(i);
		N[i] = float(i);
	}

	// allocate space for output on CPU
	float* P = (float*) malloc(size);


	// use cudaMalloc to allocate on the GPU

	// pointers for device memory
	float* Md;
	float* Nd;
	float* Pd;
	

	hipMalloc((void**)&Md, size);
	hipMalloc((void**)&Nd, size);
	hipMalloc((void**)&Pd, size);

	// transfer M and N onto the GPU's Md and Nd

	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	// invoke the kernel HERE
	// NOTE: using global memory only (cudaMalloc?), since using __shared__ was not specified in the directions
	// and this project is already difficult enough such that I'd rather not give myself extra work.
	// Also the matrices are really small, so the optimization would add extra overhead.
	// I could indeed look through the "CUDA Part 2" slides and copy over code optimized with tiles & shared memory
	// but I'd rather not for this exercise. I will not be giving myself any bonus points for effort.

	dim3 dimBlock(MAT_WIDTH, MAT_WIDTH);
	dim3 dimGrid(1,1);


	// print results. use the same P for each kernel - print before running the next operation
	for (int i = 0; i < 3; i++) {
		char* whichOp;
		switch (i) {
		case 0:
			whichOp = "ADD";
			mat_add<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
			break;
		case 1:
			whichOp = "SUBTRACT";
			mat_sub<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
			break;
		case 2:
			whichOp = "MULTIPLY";
			mat_mult<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
			break;
		default:
			//do nothing, this won't happen, just look at the for loop
			break;
		}


		// copy back onto CPU from GPU to let me print results to console
		hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
		printResults(P, whichOp);
	}

	// free device matrices
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	// junk I'll keep in case I want it later

	//for (int i = 0; i < 26; i++) {
	//	std::cout << M[i] << std::endl;
	//}

	// print out serial results
	cpu_mat_add(P, M, N);
	printResults(P, "SERIAL ADD");
	cpu_mat_sub(P, M, N);
	printResults(P, "SERIAL SUBTRACT");
	cpu_mat_mult(P, M, N);
	printResults(P, "SERIAL MULTIPLY");


	std::cin.ignore();
	return 0;
}

// just prints results
void printResults (float* R, char* message) {
	std::cout << message << std::endl;

	for (int i = 0; i < MAT_WIDTH; i++) {
		for (int j = 0; j < MAT_WIDTH; j++) {
			int index = i * MAT_WIDTH + j;
			std::cout << R[index] << '\t';
		}
		std::cout << std::endl;
	}
}

//serial versions. code is basically the same as the GPU versions, with extra nested for loops
void cpu_mat_add (float* R, float* M, float* N) {
	for (int i = 0; i < MAT_WIDTH; i++) {
		for (int j = 0; j < MAT_WIDTH; j++) {
			int index = i*MAT_WIDTH + j;
			R[index] = M[index] + N[index];
		}
	}
}

void cpu_mat_sub (float* R, float* M, float* N) {
	for (int i = 0; i < MAT_WIDTH; i++) {
		for (int j = 0; j < MAT_WIDTH; j++) {
			int index = i*MAT_WIDTH + j;
			R[index] = M[index] - N[index];
		}
	}
}

void cpu_mat_mult (float* R, float* M, float* N) {
	for (int i = 0; i < MAT_WIDTH; i++) {
		for (int j = 0; j < MAT_WIDTH; j++) {
			int index = i*MAT_WIDTH + j;
			
			float runningSum = 0.f;
			
			for (int k = 0; k < MAT_WIDTH; k++) {
				runningSum += M[i*MAT_WIDTH + k] * N[k*MAT_WIDTH + j];
			}
			
			R[index] = runningSum;
		}
	}
}
