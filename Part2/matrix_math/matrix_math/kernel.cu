#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size);
void serialMat_Mult(const float *a, const float *b, float *c, unsigned int width);
void serialMat_Add(const float *a, const float *b, float *c, unsigned int width);
void serialMat_Sub(const float *a, const float *b, float *c, unsigned int width);

__global__ void mat_add (const float *Md, const float *Nd, float *c, int width)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
    c[j*width + i] = Md[j*width + i] + Nd[j*width + i];
}

__global__ void mat_sub (const float *Md, const float *Nd,float *c,  int width)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
    c[j*width + i] = Md[j*width + i] - Nd[j*width + i];
}

__global__ void mat_mult (const float *Md, const float *Nd, float *c, int width)
{
    int i = threadIdx.x;
	int j = threadIdx.y;


	float value = 0;
	for(int k = 0; k < width; ++k){

		float MdElement = Md[j * width + k];
		float NdElement = Nd[k * width + i];

		value +=  MdElement * NdElement;

	}
	c[j*width + i] = value;

}

int main()
{
    const int arraySize = 25;
	const float a[arraySize] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24};
	const float b[arraySize] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24};
    float c[arraySize] = { 0 };

   //const int arraySize = 5;
   //const float a[arraySize][arraySize] = { {1, 2, 3, 4, 6},    {6, 1, 5, 3, 8},    {2, 6, 4, 9, 9},    {1, 3, 8, 3, 4},    {5, 7, 8, 2, 5} };
   //const float b[arraySize][arraySize] = { {3, 5, 0, 8, 7},    {2, 2, 4, 8, 3},    {0, 2, 5, 1, 2},    {1, 4, 0, 5, 1},    {3, 4, 8, 2, 3} };
   // float c[arraySize][arraySize] = { 0 };


	matrixMultiply(a, b, c, 5);


    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(a, b, c, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    printf("�u1, 2, 3, 4, 6�U   �u3, 5, 0, 8, 7�U\n"
		   "�U6, 1, 5, 3, 8�U   �U2, 2, 4, 8, 3�U\n"
	       "�U2, 6, 4, 9, 9�U + �U0, 2, 5, 1, 2�U\n"
	       "�U1, 3, 8, 3, 4�U   �U1, 4, 0, 5, 1�U\n"
	       "�U5, 7, 8, 2, 5�v   �U3, 4, 8, 2, 3�v\n"
		   "= \n{%.2f,%.2f,%.2f,%.2f,%.2f}\n{%.2f,%.2f,%.2f,%.2f,%.2f}\n{%.2f,%.2f,%.2f,%.2f,%.2f}\n{%.2f,%.2f,%.2f,%.2f,%.2f}\n{%.2f,%.2f,%.2f,%.2f,%.2f}", 
		   c[0], c[1], c[2], c[3], c[4], 
		   c[5], c[6], c[7], c[8], c[9],
		   c[10], c[11], c[12], c[13], c[14], 
		   c[15], c[16], c[17], c[18], c[19],
		   c[20], c[21], c[22], c[23], c[24]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

	int i;
	scanf("%d", &i);
    return 0;
}


void serialMat_Mult(const float *a, const float *b, float *c, unsigned int width){
	for(int i = 0; i < width; ++i){
		for(int j = 0; j < width; ++j){
			float sum = 0;
			for(int k = 0 ;k < width; ++k){
				float m = a[i * width + k];
				float n = b[k * width + j];
				sum += m*n;
			}
			c[i*width+j] = sum;
		}
	}
}

void serialMat_Add(const float *a, const float *b, float *c, unsigned int width){
	for(int i = 0; i < width; ++i){
		for(int j = 0; j < width; ++j){
			c[i*width+j] = a[i*width+j] + b[i*width+j];
		}
	}
}

void serialMat_Sub(const float *a, const float *b, float *c, unsigned int width){
	for(int i = 0; i < width; ++i){
		for(int j = 0; j < width; ++j){
			c[i*width+j] = a[i*width+j] - b[i*width+j];
		}
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(const float *a, const float *b, float *c, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

	//size = 25;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	dim3 threadPerBlock(5, 5);
    mat_mult <<<1, threadPerBlock>>>(dev_a, dev_b, dev_c, 5);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
