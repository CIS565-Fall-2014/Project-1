#include <hip/hip_runtime.h>

#include <iostream>

__global__ void dev_matrix_add(int dim, float * A, float * B, float * result)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(row < dim && col < dim) result[row * dim + col] = A[row * dim + col] + B[row * dim + col];
}

__global__ void dev_matrix_sub(int dim, float * A, float * B, float * result)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(row < dim && col < dim) result[row * dim + col] = A[row * dim + col] - B[row * dim + col];
}

__global__ void dev_matrix_mult(int dim, float * A, float * B, float * result)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(row >= dim || col >= dim) return;
	float sum = 0.0f;
	for (int i = 0; i < dim; i++)
	{
		sum += A[row * dim + i] * B[col * dim + i];
	}

	result[row * dim + col] = sum;
}


int main(int argc, char** argv)
{
	float * A, *B, * result, * dev_A, * dev_B, * dev_result;
	int M(9);
	int tileWidth = 2;
	int N = M * M * sizeof(float);

	A = (float*) malloc( N);
	B = (float*) malloc(N);
	result = (float*) malloc(N);

	hipMalloc((void**) & dev_A, N);
	hipMalloc((void**) & dev_B, N);
	hipMalloc((void**) & dev_result,N);

	for(int i = 0;i<M*M;i++)
	{
		A[i] = 1.0f;
		B[i] = 1.0f;
	}

	hipMemcpy(dev_A,A,N,hipMemcpyHostToDevice);
	hipMemcpy(dev_B,B,N,hipMemcpyHostToDevice);

	dim3 gridDim((int)ceil((float)M/(float)tileWidth),(int)ceil((float)M/(float)tileWidth));
	dim3 blockDim(tileWidth,tileWidth);

	dev_matrix_mult<<<gridDim,blockDim>>>(M,dev_A,dev_B,dev_result);


	hipMemcpy(result, dev_result,N,hipMemcpyDeviceToHost);
	for(int i = 0;i<M*M;i++)
	{
		std::cout<<result[i]<<std::endl;
	}
    std::cout<<"test";
	std::cin.get();
    return 0;
}

